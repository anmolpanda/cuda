#include "hip/hip_runtime.h"
/* Host side code that calls a GPU kernel to perform vector addition on the GPU using a single thread block. 
	We restrict the size of the vector to be up to 512 elements which is the maximum thread block size on this 
	GPU.

	Author: Naga Kandasamy
	Date last modified: 10/20/2014
*/


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <vector_addition_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void run_test(void);
void compute_on_device(float *, float *, float *, int);
extern "C" void compute_gold( float *, float *, float *, int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
	run_test();
	return 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Perform vector addition on the CPU and the GPU and compare results for correctness
////////////////////////////////////////////////////////////////////////////////
void run_test(void) {
	int num_elements = NUM_ELEMENTS;
	float diff;
	int i; 

	// Obtain the vector length
	int vector_length = sizeof(float) * num_elements;

    // Allocate memory on the CPU for the input vectors A and B, and the output vector C
	float *A = (float *)malloc(vector_length);
	float *B = (float *)malloc(vector_length);
	float *gold_result = (float *)malloc(vector_length); // The result vector computed on the CPU
	float *gpu_result = (float *)malloc(vector_length); // The result vector computed on the GPU
	
	// Randomly generate input data. Initialize the input data to be integer values between 0 and 100 
	for(i = 0; i < num_elements; i++){
		A[i] = floorf(100*(rand()/(float)RAND_MAX));
     	B[i] = floorf(100*(rand()/(float)RAND_MAX));
	}

	// Compute the reference solution on the CPU
	compute_gold(A, B, gold_result, num_elements);
    
	// Compute the result vector on the GPU 
	compute_on_device(A, B, gpu_result, num_elements);

	// Compute the differences between the CPU and GPU results
	diff = 0.0;
   for(i = 0; i < num_elements; i++)
		diff = diff + abs(gold_result[i] - gpu_result[i]);

	printf("Difference between the CPU and GPU result: %f. \n", diff);
   
	// cleanup memory
	free(A);
	free(B);
	free(gold_result);
	free(gpu_result);
	
	return;
}

// Transfer vectors A and B from the CPU to the GPU, setup grid and thread dimentions, excute kernel function, and copy result vector back to the CPU
void compute_on_device(float *A_on_host, float *B_on_host, float *gpu_result, int num_elements){
	float *A_on_device = NULL;
	float *B_on_device = NULL;
	float *C_on_device = NULL; 

	// Allocate space on the GPU for vectors A and B, and copy the contents of the vectors to the GPU
	hipMalloc((void**)&A_on_device, num_elements*sizeof(float));
	hipMemcpy(A_on_device, A_on_host, num_elements*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&B_on_device, num_elements*sizeof(float));
	hipMemcpy(B_on_device, B_on_host, num_elements*sizeof(float), hipMemcpyHostToDevice);

	// Allocate space for the result vector on the GPU
	hipMalloc((void**)&C_on_device, num_elements*sizeof(float));
	
 	// Set up the execution grid on the GPU 
	dim3 thread_block(num_elements, 1, 1); // Set the number of threads in the thread block
	dim3 grid(1,1);
	vector_addition_kernel<<<grid, thread_block>>>(A_on_device, B_on_device, C_on_device, num_elements); // Launch the kernel with a single thread block

	// Copy the result vector back from the GPU and store 
	hipMemcpy(gpu_result, C_on_device, num_elements*sizeof(float), hipMemcpyDeviceToHost);
	
	// Free memory on the GPU
	hipFree(A_on_device);
	hipFree(B_on_device);
	hipFree(C_on_device);
}
     
